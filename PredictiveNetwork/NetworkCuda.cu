#include "hip/hip_runtime.h"
#include "NetworkCuda.cuh"
#include <hip/hip_runtime.h>
#include ""
#define GPU_ERROR_RET(msg, err) if(err != hipSuccess) { std::cerr << "\n>> " __FILE__ " at line " << __LINE__ << ":\n<< " #msg << ": " << hipGetErrorString(err) << std::endl; return false; }
#define GPU_ERROR_ABT(msg, err) if(err != hipSuccess) { std::cerr << "\n>> " __FILE__ " at line " << __LINE__ << ":\n<< " #msg << ": " << hipGetErrorString(err) << std::endl; abort(); }
__global__ void updateVector(
	const float* vector, 
	const float* matrix,
	float* output,
	const size_t sizeV, 
	const size_t sizeM,
	const size_t sizeO
) {

	__shared__ float sums[1024u];

	sums[threadIdx.x] = 0.0f;

	//Global IDs
	uint32_t vectorID1 = threadIdx.x;
	uint32_t matrixID1 = sizeV * blockIdx.x + threadIdx.x;

	while (vectorID1 < sizeV && matrixID1 < sizeM) {
		sums[threadIdx.x] += vector[vectorID1] * matrix[matrixID1];
		vectorID1 += blockDim.x;
		matrixID1 += blockDim.x;
	}

	__syncthreads();
	if (threadIdx.x < 512u) sums[threadIdx.x] += sums[threadIdx.x + 512u];
	__syncthreads();
	if (threadIdx.x < 256u) sums[threadIdx.x] += sums[threadIdx.x + 256u];
	__syncthreads();
	if (threadIdx.x < 128u) sums[threadIdx.x] += sums[threadIdx.x + 128u];
	__syncthreads();
	if (threadIdx.x < 64u) sums[threadIdx.x] += sums[threadIdx.x + 64u];
	__syncthreads();
	if (threadIdx.x < 32u) sums[threadIdx.x] += sums[threadIdx.x + 32u];
	__syncthreads();
	if (threadIdx.x < 16u) sums[threadIdx.x] += sums[threadIdx.x + 16u];
	__syncthreads();
	if (threadIdx.x < 8u) sums[threadIdx.x] += sums[threadIdx.x + 8u];
	__syncthreads();
	if (threadIdx.x < 4u) sums[threadIdx.x] += sums[threadIdx.x + 4u];
	__syncthreads();
	if (threadIdx.x < 2u) sums[threadIdx.x] += sums[threadIdx.x + 2u];
	__syncthreads();
	if (threadIdx.x < 1u) sums[threadIdx.x] += sums[threadIdx.x + 1u];
	__syncthreads();
	output[blockIdx.x] = sums[0u];
}

NetworkCuda::NetworkCuda():
values(nullptr),
errors(nullptr),
weightsV(nullptr),
weightsE(nullptr)
{
	/*hipMalloc((void**)&values, getVectorSize() * sizeof(float));
	hipMalloc((void**)&errors, getVectorSize() * sizeof(float));
	hipMalloc((void**)&weightsV, getMatrixSize() * sizeof(float));
	hipMalloc((void**)&weightsE, getMatrixSize() * sizeof(float));*/
};

NetworkCuda::~NetworkCuda() {
	/*hipFree(values);
	hipFree(errors);
	hipFree(weightsV);
	hipFree(weightsE);*/
};

void NetworkCuda::testMatrixMultiplication() {
	const static uint32_t sizesV = 1024u;
	const static uint32_t sizesM = sizesV * sizesV;

	float* testVector_d = nullptr;
	float* testMatrix_d = nullptr;
	float* testResults_d = nullptr;

	float* testVector_h = nullptr;
	float* testMatrix_h = nullptr;
	float* testResults_h = nullptr;

	float* testResults_dh = nullptr;

	testVector_h = new float[sizesV];
	testMatrix_h = new float[sizesM];
	testResults_h = new float[sizesV];
	testResults_dh = new float[sizesV];

	std::srand(334);

	for (uint32_t u = 0u; u < sizesV; u++) {
		testVector_h[u] = ((float)std::rand() / (float)RAND_MAX);
		testResults_h[u] = 0u;
		for (uint32_t v = 0u; v < sizesV; v++) {
			testMatrix_h[v + (sizesV * u)] = ((float)std::rand() / (float)RAND_MAX);
		}
	}
	hipMalloc((void**)&testVector_d, sizesV * sizeof(float));
	hipMalloc((void**)&testMatrix_d, sizesM * sizeof(float));
	hipMalloc((void**)&testResults_d, sizesV * sizeof(float));
	hipMemcpy(testVector_d, testVector_h, sizesV * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(testMatrix_d, testMatrix_h, sizesM * sizeof(float), hipMemcpyHostToDevice);
	for (uint32_t u = 0u; u < sizesV; u++) {
		for (uint32_t v = 0u; v < sizesV; v++) {
			testResults_h[u] += testVector_h[v] * testMatrix_h[(sizesV * u) + v];
		}
	}

	delete[] testVector_h;
	delete[] testMatrix_h;

	updateVector<<<sizesV,(sizesV > 1024u) ? 1024u : sizesV>>>(testVector_d, testMatrix_d, testResults_d, sizesV, sizesM, sizesV);
	hipDeviceSynchronize();
	hipMemcpy(testResults_dh, testResults_d, sizesV * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(testVector_d);
	hipFree(testMatrix_d);
	hipFree(testResults_d);
	float score = 0.0f;
	for (uint32_t u = 0u; u < sizesV; u++) {
		float sum = testResults_h[u] - testResults_dh[u];
		score += sum * sum;
	}

	delete[] testResults_h;
	delete[] testResults_dh;
	
	std::cout << "Matrix Kernel Test Results: " << score << '\n';
};